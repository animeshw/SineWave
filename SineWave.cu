#include "hip/hip_runtime.h"
#include "SineWave.cuh"

__global__ void sine_wave_kernel(float4* pos, unsigned int mesh_width, unsigned int mesh_height, float time) {

    //Code
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float u = x / (float)mesh_width;
    float v = y / (float)mesh_height;
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;
    float frequency = 4.0f;
    float w = sinf(u * frequency + time) * cosf(v * frequency + time) * 0.5f;

    pos[y * mesh_width + x] = make_float4(u, w, v, 1.0f);
}

void launch_cuda_kernel(float4* pos, unsigned int mesh_width, unsigned int mesh_height, float time) {

    //Code
    dim3 block = dim3(8, 8, 1);
    dim3 grid = dim3(mesh_width / block.x, mesh_height / block.y, 1);

    sine_wave_kernel << <grid, block >> > (pos, mesh_width, mesh_height, time);
}
